#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda/atomic>
#include <chrono> // Per misurare il tempo

#define M 1024
#define N 1024
#define NUM_BLOCKS 1024
#define NUM_THREADS_PER_BLOCK 1024

//optimistic approach
__global__ void find_minimum_test(int *input, int *global_minimum);
__global__ void find_minimum_fix(int *input, int *global_minimum);
__global__ void find_minimum_opt(int *input, int *global_minimum);




int main() {
    
    // Get a different random number each time the program runs
    srand(time(0));
    hipError_t err;

    
    // Declare and initialize global minimum on CPU
    int global_minimum_cpu[1]; global_minimum_cpu[0] = INT_MAX; std::cout << "Initial max number: " << global_minimum_cpu[0] << std::endl;
    
    
    // Allocate memory for global minimum on GPU; two variables: one for each approach
    int *global_minimum_gpu_opt; int *global_minimum_gpu_fix;
    hipMalloc(&global_minimum_gpu_opt, sizeof(int)); hipMalloc(&global_minimum_gpu_fix, sizeof(int));
    //The follwing lines can be useful to detect errors; If there is an error with the CUDA operation we get -1
    err = hipGetLastError();
    if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       
        // Possibly: exit(-1) if program cannot continue....
    }

    //we assign the initial max value to the variables of gpu
    hipMemcpy(global_minimum_gpu_opt, global_minimum_cpu, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(global_minimum_gpu_fix, global_minimum_cpu, sizeof(int), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       
    }
    
    // Declare and initialize input array on CPU; this is just an example; the input array can be whatever
    int input_array_cpu[M * N];
    for (int i = 0; i < M * N; i++) {
        input_array_cpu[i] = i + 2; 
    }

    // Allocate memory for input array on GPU
    int *input_array_gpu;
    hipMalloc(&input_array_gpu, M * N * sizeof(int));
        err = hipGetLastError();
    if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(input_array_gpu, input_array_cpu, M * N * sizeof(int), hipMemcpyHostToDevice); 
    err = hipGetLastError();
    if ( err != hipSuccess )
     {
        printf("here CUDA Error: %s\n", hipGetErrorString(err));       
    }
    hipDeviceSynchronize();

    //We loop and we a lot of time and aunch kernel; at the end we converge and we get the minimum
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 1000000; i++) {

        find_minimum_opt<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(input_array_gpu, global_minimum_gpu_opt);
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }
        hipDeviceSynchronize();

    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    // Copy result back to CPU
    hipMemcpy(global_minimum_cpu, global_minimum_gpu_opt, sizeof(int), hipMemcpyDeviceToHost); hipDeviceSynchronize();
    // Output the result
    std::cout << "Computed minimum value by the optimistic approach: " << global_minimum_cpu[0] << "  , in "<< duration.count() << "  seconds" <<std::endl;
    
    
    start = std::chrono::high_resolution_clock::now();
    find_minimum_fix<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(input_array_gpu, global_minimum_gpu_fix);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    hipMemcpy(global_minimum_cpu, global_minimum_gpu_opt, sizeof(int), hipMemcpyDeviceToHost); hipDeviceSynchronize();
    std::cout << "Computed minimum value by fix point approach: " << global_minimum_cpu[0] << "  , in "<< duration.count() << "  seconds" <<std::endl;

    //The following is just a sligtly different implementation which exploit the synchronization between threads to find the minimum
    //I left it on purpose because I think it can be interesting
    start = std::chrono::high_resolution_clock::now();
    find_minimum_test<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(input_array_gpu, global_minimum_gpu_fix);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    hipMemcpy(global_minimum_cpu, global_minimum_gpu_opt, sizeof(int), hipMemcpyDeviceToHost); hipDeviceSynchronize();
    std::cout << "Computed minimum value by test: " << global_minimum_cpu[0] << "  , in "<< duration.count() << "  seconds" <<std::endl;

    // Free GPU memory
    hipFree(global_minimum_gpu_opt);hipFree(global_minimum_gpu_fix);
    hipFree(input_array_gpu);
}

//The following is just a sligtly different implementation which exploit the synchronization between threads to find the minimum
//I left it on purpose because I think it can be interesting
__global__ void find_minimum_test(int *input, int *global_minimum) {
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    for(int i = 0; i < 100000; i++){
        if (input[index] < global_minimum[0])
            global_minimum[0] = input[index];
        __syncthreads();
    }
    //atomicExch(global_minimum, input[index]);
}
//fix approach
__global__ void find_minimum_fix(int *input, int *global_minimum) {
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    if (input[index] < global_minimum[0])
        atomicExch(global_minimum, input[index]);
}


//optimistic approach
__global__ void find_minimum_opt(int *input, int *global_minimum) {
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    if (input[index] < global_minimum[0]) 
        global_minimum[0] = input[index]; 
}
