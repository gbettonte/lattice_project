#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda/atomic>
#include <chrono> // Per misurare il tempo

#define M 1024
#define N 1024
#define NUM_BLOCKS 1024
#define NUM_THREADS_PER_BLOCK 1024

//optimistic approach
__global__ void find_minimum_test(int *input, int *global_minimum);
__global__ void find_minimum_fix(int *input, int *global_minimum);
__global__ void find_minimum_opt(int *input, int *global_minimum);




int main() {
    
    // Get a different random number each time the program runs
    srand(time(0));
    hipError_t err;

    
    // Declare and initialize global minimum on CPU
    int global_minimum_cpu[1]; global_minimum_cpu[0] = INT_MAX; std::cout << "Initial max number: " << global_minimum_cpu[0] << std::endl;
    
    
    // Allocate memory for global minimum on GPU
    int *global_minimum_gpu_opt; int *global_minimum_gpu_fix;
    hipMalloc(&global_minimum_gpu_opt, sizeof(int)); hipMalloc(&global_minimum_gpu_fix, sizeof(int)); 
    err = hipGetLastError();
    if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       

        // Possibly: exit(-1) if program cannot continue....
    }

    hipMemcpy(global_minimum_gpu_opt, global_minimum_cpu, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(global_minimum_gpu_fix, global_minimum_cpu, sizeof(int), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       

        // Possibly: exit(-1) if program cannot continue....
    }
    
    // Declare and initialize input array on CPU
    int input_array_cpu[M * N];
    for (int i = 0; i < M * N; i++) {
        input_array_cpu[i] = i + 2; 
    }

    // Allocate memory for input array on GPU
    int *input_array_gpu;
    hipMalloc(&input_array_gpu, M * N * sizeof(int));
        err = hipGetLastError();
    if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       

        // Possibly: exit(-1) if program cannot continue....
    }
    hipMemcpy(input_array_gpu, input_array_cpu, M * N * sizeof(int), hipMemcpyHostToDevice); 
    
    
        err = hipGetLastError();
    if ( err != hipSuccess )
     {
        printf("here CUDA Error: %s\n", hipGetErrorString(err));       

        // Possibly: exit(-1) if program cannot continue....
    }
    hipDeviceSynchronize();

    // Launch kernel
    //find_minimum<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(input_array_gpu, global_minimum_gpu);
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 1000000; i++) {

        find_minimum_opt<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(input_array_gpu, global_minimum_gpu_opt);
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       

            // Possibly: exit(-1) if program cannot continue....
        }
        hipDeviceSynchronize();

    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    // Copy result back to CPU
    hipMemcpy(global_minimum_cpu, global_minimum_gpu_opt, sizeof(int), hipMemcpyDeviceToHost); hipDeviceSynchronize();
    // Output the result
    std::cout << "Computed minimum value by opt: " << global_minimum_cpu[0] << "  , in "<< duration.count() << "  seconds" <<std::endl;
    
    
    start = std::chrono::high_resolution_clock::now();
    find_minimum_fix<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(input_array_gpu, global_minimum_gpu_fix);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    hipMemcpy(global_minimum_cpu, global_minimum_gpu_opt, sizeof(int), hipMemcpyDeviceToHost); hipDeviceSynchronize();
    std::cout << "Computed minimum value by fix: " << global_minimum_cpu[0] << "  , in "<< duration.count() << "  seconds" <<std::endl;

    start = std::chrono::high_resolution_clock::now();
    find_minimum_test<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(input_array_gpu, global_minimum_gpu_fix);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    hipMemcpy(global_minimum_cpu, global_minimum_gpu_opt, sizeof(int), hipMemcpyDeviceToHost); hipDeviceSynchronize();
    std::cout << "Computed minimum value by test: " << global_minimum_cpu[0] << "  , in "<< duration.count() << "  seconds" <<std::endl;

    // Free GPU memory
    hipFree(global_minimum_gpu_opt);hipFree(global_minimum_gpu_fix);
    hipFree(input_array_gpu);
}

__global__ void find_minimum_test(int *input, int *global_minimum) {
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    for(int i = 0; i < 100000; i++){
        if (input[index] < global_minimum[0])
            global_minimum[0] = input[index];
        __syncthreads();
    }
    //atomicExch(global_minimum, input[index]);
}
//optimistic approach
__global__ void find_minimum_fix(int *input, int *global_minimum) {
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    if (input[index] < global_minimum[0])
        atomicExch(global_minimum, input[index]);
}


//optimistic approach
__global__ void find_minimum_opt(int *input, int *global_minimum) {
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    if (input[index] < global_minimum[0]) 
        global_minimum[0] = input[index]; 
}
