#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>
#include <cstring>
#include <utility>
#include <vector>
#include <chrono>
#include <stack>
#include <set>
#include <algorithm>
#include "parser.hpp"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <cuda/atomic>
#define NUM_BLOCKS 1024
#define NUM_THREADS_PER_BLOCK 1024
using namespace std;




struct Node {
  int depth; // depth in the tree
  std::vector<int> domains;

  Node(size_t N, Data data): depth(0), domains(){
    for (int i = 0; i < N; i++) {
      int max = data.get_max_u();
      domains.resize(N * max);
      for(int j = 0; j < max; j++)
        domains[i*max+j] = 1;
    }
  }

  Node(const Node&) = default; //allow to copy an existing node
  Node(Node&&) = default;      //allow to move a node
  Node() = default; 
};



//CUDA version of the update domanis; very similar to the OpenMP implementation
//here each thread retrieves its index; if it can it manages an iteration of the loop
__global__ void update_domains_cuda(int* domains, int* parent_depth, int* n, int* j, int* array_C, int* max_u) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int value_n = *n;
    int column = *j;
    int max = *max_u;
    int starting_depth = *parent_depth + 1;

    if (index < value_n  && index >= starting_depth) {
        int idx = index * max + column;
        if (array_C[idx] == 1) {
          domains[idx] = 0;
        }
    }
}

//This version of update_domanins uses OpenMP; each thread manages an iteration of the loop
//(this function is not used in this code)
void update_domains(std::vector<int>& domains, int parent_depth, int starting_depth, int j, int n, int max_u, int* u, int* array_C){
#pragma omp parallel for num_threads(4)  
  for(int i = starting_depth; i < n; i++){
    if( array_C[i*max_u+j] == 1)
      domains[i*max_u+j] = 0;
  }
  return;
}



// evaluate a given node and branch it if it is valid
void evaluate_and_branch(const Node& parent, std::stack<Node>& pool, size_t& tree_loc, size_t& num_sol, int n, int max_u, int* u,  int* array_C)
{
  int depth = parent.depth;
  // if the given node is a leaf, then update counter and do nothing
  if (depth == n) {
    num_sol++;
  }
  // if the given node is not a leaf, then update counter and evaluate/branch it
  else{
    int upper_bound = u[depth];
    for(int j = 0; j < upper_bound; j++){
      if(parent.domains[depth*max_u + j] == 1){
        //call update domains
        Node child(parent);
        child.depth++;
        tree_loc++;

        //update_domains(child.domains, parent.depth, child.depth, j, n, max_u, u, array_C);

        //before call update_domains_cuda is necessary to transfer the necessary struct to the gpu
        //child domains
        int *child_domains_gpu; hipMalloc(&child_domains_gpu, n* max_u * sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(child_domains_gpu, &child.domains, n * max_u * sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //array_C
        int *array_C_gpu; hipMalloc(&array_C_gpu, n * max_u *sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(array_C_gpu, array_C, n*n*sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //parent.depth&
        int *parent_depth_gpu; hipMalloc(&parent_depth_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(parent_depth_gpu, &(parent.depth), sizeof(int), hipMemcpyHostToDevice);hipDeviceSynchronize();

        //child.depth
        int *child_depth_gpu; hipMalloc(&child_depth_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(child_depth_gpu, &(child.depth), sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //j (column)
        int *j_gpu; hipMalloc(&j_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(j_gpu, &j, sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //n
        int *n_gpu; hipMalloc(&n_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(n_gpu, &n, sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        int *max_u_gpu; hipMalloc(&max_u_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(max_u_gpu, &max_u, sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();
        //update_domains_cuda<<< NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(child_domains_gpu, parent_depth_gpu, child_depth_gpu, j_gpu, data_gpu); 

        //now we can call update_domains_cuda
        //__global__ void update_domains_cuda(bool *domains, int *parent_depth, int* n, int *starting_depth, int *j, Data *data )
        update_domains_cuda<<<1, 10>>>(child_domains_gpu, parent_depth_gpu, n_gpu, j_gpu, array_C_gpu, max_u_gpu);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
          printf("Error launch kernel: %s\n", hipGetErrorString(err));
        }hipDeviceSynchronize();
        //and then copy back to CPU
        //child domains //we just need to copy back what was actually modified
        hipMemcpy(&child.domains, child_domains_gpu, n * max_u * sizeof(bool), hipMemcpyDeviceToHost); hipDeviceSynchronize();
        

        //finally we push the child into the stack
        pool.push(std::move(child));
      }
    }
  }
}



int main(int argc, char** argv) {
  Data data;
  
  //print some information
  if (data.read_input("pco_3.txt")){
    data.print_n();
    data.print_u();
    data.print_C();
  }
    

  int* u = data.get_u();
  int n = data.get_n();
  int max_u = data.get_max_u();
  int** C = data.get_C(); //we retrieve the constraints matrix
  int* array_C = (int*)malloc(n* n * sizeof(int)); // we allocate a 1D array (to simplify cuda operations)
  
  //we print the constrainta matrix
  int i, j;
  for(i = 0; i < n; i++)
    std::cout << u[i];
    std::cout << endl;
    std::cout << endl;
    for(i = 0; i < n; i++){
      for(j = 0; j < n; j++){
        array_C[i * n + j] = C[i][j];
        std ::cout<< array_C[i * n + j] << " ";
      }
      std::cout << endl;
    }

  // helper
  if (argc != 2) {
    std::cout << "usage: " << argv[0] << " <number of queens> " << std::endl;
    exit(1);
  }

  // problem size (number of variables)
  size_t N = std::stoll(argv[1]);
  std::cout << "Solving " << N << "-Queens problem\n" << std::endl;

  // initialization of the root node (the board configuration where no queen is placed)
  Node root(N, data);

  // initialization of the pool of nodes (stack -> DFS exploration order)
  std::stack<Node> pool; //stack of nodes; a stack is LIFO First IN First OUT
  pool.push(std::move(root)); //push the root on the stack

  // statistics to check correctness (number of nodes explored and number of solutions found)
  size_t exploredTree = 0;
  size_t exploredSol = 0;

  // beginning of the Depth-First tree-Search
  auto start = std::chrono::steady_clock::now();

  while (pool.size() != 0) { //i.e continue till all the the path are explored
    // get a node from the pool
    Node currentNode(std::move(pool.top()));
    pool.pop();

    // check the board configuration of the node and branch it if it is valid.
    evaluate_and_branch(currentNode, pool, exploredTree, exploredSol, n, max_u, u, array_C);


  }

  //get the finish time
  auto end = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

  // outputs
  std::cout << "Time taken with CUDA: " << duration.count() << " milliseconds" << std::endl;
  std::cout << "Total solutions: " << exploredSol << std::endl;
  //std::cout << "Size of the explored tree: " << exploredTree << std::endl;

  return 0;
}
/*
 * Author: Guillaume HELBECQUE (Université du Luxembourg)
 * Date: 10/10/2024
 *
 * Description:
 * This program solves the N-Queens problem using a sequential Depth-First tree-Search
 * (DFS) algorithm. It serves as a basis for task-parallel implementations.
 */

#include <omp.h>

#include <iostream>
#include <cstring>
#include <utility>
#include <vector>
#include <chrono>
#include <stack>
#include <set>
#include <algorithm>
#include "parser.hpp"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda/atomic>
#include <chrono>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda/atomic>
#define NUM_BLOCKS 1024
#define NUM_THREADS_PER_BLOCK 1024
using namespace std;




// N-Queens node
struct Node {
  int depth; // depth in the tree
  std::vector<int> board; // board configuration (permutation)
  //Example of permutation with 4 queens: [2, 0, 3, 1] queen of the first row on the second column, and so on
  //std::set<int> possible_places;
  std::vector<int> domains;

  Node(size_t N, Data data): depth(0), board(N), domains(){
    for (int i = 0; i < N; i++) {
      board[i] = i;
      //domains.resize(N * std::vector<bool>(data.get_max_u(), true));
      int max = data.get_max_u();
      domains.resize(N * max);
      for(int j = 0; j < max; j++)
        domains[i*max+j] = 1;
    }
  }

  Node(const Node&) = default; //allow to copy an existing node
  Node(Node&&) = default;      //allow to move a node
  Node() = default; 
};

// check if placing a queen is safe (i.e., check if all the queens already placed share
// a same diagonal)
//Modify nqueens.cpp to check an arbitrary list of inequalities. 
//I slightly modify this function to take into account other constraints
//          //current configuration         //row anc column where we have to evaluate



__global__ void update_domains_cuda(int* domains, int* parent_depth, int* n, int* j, int* array_C, int* max_u) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    int value_n = *n;
    int column = *j;
    int max = *max_u;
    int starting_depth = *parent_depth + 1;

    if (index < value_n  && index >= starting_depth) {
        int idx = index * max + column;
        printf("%d  ", idx);
        if (array_C[idx] == 1) {
          domains[idx] = 0;
        }
    }
}

//
//now use cuda to parallelize this
//update_domains(child.domains, parent.depth, child.depth, j, n, max_u, u, array_C);
void update_domains(std::vector<int>& domains, int parent_depth, int starting_depth, int j, int n, int max_u, int* u, int* array_C){
#pragma omp parallel for num_threads(4)  
  for(int i = starting_depth; i < n; i++){
    if( array_C[i*max_u+j] == 1)
      domains[i*max_u+j] = 1;
  }
  return;
}

bool isSafe(/*const std::vector<int>& board, */const int row, const int col, Data data)
{
  if(data.get_C_at(row, col) == 1)
    return false;

  return true;
}

// evaluate a given node (i.e., check its board configuration) and branch it if it is valid
// (i.e., generate its child nodes.)
//              evaluate_and_branch(currentNode, pool, exploredTree, exploredSol, n, max_u, u, array_C);
void evaluate_and_branch(const Node& parent, std::stack<Node>& pool, size_t& tree_loc, size_t& num_sol, int n, int max_u, int* u,  int* array_C)
{
  int depth = parent.depth;
  int N = parent.board.size();

  // if the given node is a leaf, then update counter and do nothing
  if (depth == N) {
    num_sol++;
  }
  // if the given node is not a leaf, then update counter and evaluate/branch it
  /*
  else{
    for (int j = depth; j < N; j++) {
      if (isSafe(parent.board, depth, parent.board[j], data)) {
        Node child(parent);
        std::swap(child.board[depth], child.board[j]);
        child.depth++;
        pool.push(std::move(child));
        tree_loc++;
      }
    }
  }
  */

  else{
    int upper_bound = u[depth];
    for(int j = 0; j < upper_bound; j++){
      if(parent.domains[depth*max_u + j] == 1){
        //call update domains
        Node child(parent);
        child.depth++;
        tree_loc++;
        //update_domains(child.domains, parent.depth, child.depth, j, n, max_u, u, array_C);
        //before call update_domains_cuda is necessary to transfer the necessary struct to the gpu

        //child domains
        int *child_domains_gpu; hipMalloc(&child_domains_gpu, n* max_u * sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(child_domains_gpu, &child.domains, n * max_u * sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //array_C
        int *array_C_gpu; hipMalloc(&array_C_gpu, n * max_u *sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(array_C_gpu, array_C, n*n*sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //parent.depth&
        int *parent_depth_gpu; hipMalloc(&parent_depth_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(parent_depth_gpu, &(parent.depth), sizeof(int), hipMemcpyHostToDevice);hipDeviceSynchronize();

        //child.depth
        int *child_depth_gpu; hipMalloc(&child_depth_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(child_depth_gpu, &(child.depth), sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //j (column)
        int *j_gpu; hipMalloc(&j_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(j_gpu, &j, sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //n
        int *n_gpu; hipMalloc(&n_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(n_gpu, &n, sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();

        int *max_u_gpu; hipMalloc(&max_u_gpu, sizeof(int)); hipDeviceSynchronize();
        hipMemcpy(max_u_gpu, &max_u, sizeof(int), hipMemcpyHostToDevice); hipDeviceSynchronize();
        //update_domains_cuda<<< NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(child_domains_gpu, parent_depth_gpu, child_depth_gpu, j_gpu, data_gpu); 

        //data
        //Data *data_gpu; hipMalloc(&data_gpu, sizeof(Data)); hipDeviceSynchronize();
        //hipMemcpy(data_gpu, &data, sizeof(Data), hipMemcpyHostToDevice); hipDeviceSynchronize();

        //now we can call update_domains_cuda
        //__global__ void update_domains_cuda(bool *domains, int *parent_depth, int* n, int *starting_depth, int *j, Data *data )
        update_domains_cuda<<<1, 10>>>(child_domains_gpu, parent_depth_gpu, n_gpu, j_gpu, array_C_gpu, max_u_gpu);
        std::cout << "here!";
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
          printf("Errore lancio kernel: %s\n", hipGetErrorString(err));
        }hipDeviceSynchronize();
        //and then copy back to CPU
        //child domains //we just need to copy back what was actually modified
        hipMemcpy(&child.domains, child_domains_gpu, n * max_u * sizeof(bool), hipMemcpyDeviceToHost); hipDeviceSynchronize();
        

        //finally we push the child into the stack
        pool.push(std::move(child));
      }
      

    }
  }
  /*
  else {
    for (auto it = parent.possible_places.begin(); it != parent.possible_places.end(); ) {
      if (isSafe(depth, *it, data)) {
          parent.possible_places.erase(it++);
          parent.possible_places.erase(it++);
          Node child(parent);
          child.depth++;
          pool.push(std::move(child));
          tree_loc++;
      }
      else {
        ++it;
      }
  }
}
*/
  }
  
 /*
 //my version with backtracking
  else{
    for(int j = 0; j < parent.possible_places[depth].size(); j++){
      if (parent.possible_places[depth][j] == -1) {
        continue; // Skip invalid positions
      }
      Node child(parent); child.depth++; tree_loc++; pool.push(std::move(child));
      if(depth + 1 != N){
        for(int k = depth + 1; k < N; k++){
          if(data.get_C_at(k, depth) == 1){
            child.possible_places[k][j] = -1;

          }
        }
      }
  }
}
*/


int main(int argc, char** argv) {
    Data data;
    
    if (data.read_input("pco_3.txt")){
        data.print_n();
        data.print_u();
        data.print_C();
    }
    

  //test print
  //inline int get_u_at(size_t i){return u[i];}
  //std::cout << "u[0]:  " <<data.get_u_at(0) << std::endl;
  //std::cout << "MAX:  " <<data.get_max_u() << std::endl;
  //get useful and constant information
  int* u = data.get_u();
  int n = data.get_n();
  int max_u = data.get_max_u();
  int** C = data.get_C(); //we retrieve the constraints matrix
  int* array_C = (int*)malloc(n* n * sizeof(int)); // we allocate a 1D array
  
  int i, j;
  for(i = 0; i < n; i++)
    std::cout << u[i];
  std::cout << endl;
  std::cout << endl;
  for(i = 0; i < n; i++){
    for(j = 0; j < n; j++){
      array_C[i * n + j] = C[i][j];
      std ::cout<< array_C[i * n + j] << " ";
    }
    std::cout << endl;
  }

  // helper
  if (argc != 2) {
    std::cout << "usage: " << argv[0] << " <number of queens> " << std::endl;
    exit(1);
  }

  // problem size (number of queens)
  size_t N = std::stoll(argv[1]);
  std::cout << "Solving " << N << "-Queens problem\n" << std::endl;

  // initialization of the root node (the board configuration where no queen is placed)
  Node root(N, data);

  // initialization of the pool of nodes (stack -> DFS exploration order)
  std::stack<Node> pool; //stack of nodes; a stack is LIFO First IN First OUT
  pool.push(std::move(root)); //push the root on the stack

  // statistics to check correctness (number of nodes explored and number of solutions found)
  size_t exploredTree = 0;
  size_t exploredSol = 0;

  // beginning of the Depth-First tree-Search
  auto start = std::chrono::steady_clock::now();

  while (pool.size() != 0) { //i.e continue till all the the path are explored
    // get a node from the pool
    Node currentNode(std::move(pool.top()));
    pool.pop();

    // check the board configuration of the node and branch it if it is valid.
    evaluate_and_branch(currentNode, pool, exploredTree, exploredSol, n, max_u, u, array_C);


  }

  //get the finish time
  auto end = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

  // outputs
  std::cout << "Time taken: " << duration.count() << " milliseconds" << std::endl;
  std::cout << "Total solutions: " << exploredSol << std::endl;
  std::cout << "Size of the explored tree: " << exploredTree << std::endl;

  return 0;
}
